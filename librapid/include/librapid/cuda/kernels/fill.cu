#include "hip/hip_runtime.h"
// MT19937 RNG from https://github.com/bstatcomp/RandomCL/blob/master/generators/mt19937.cl

/**
@file

Implements Mersenne twister generator.

  M. Matsumoto, T. Nishimura, Mersenne twister: a 623-dimensionally equidistributed uniform
pseudo-random number generator, ACM Transactions on Modeling and Computer Simulation (TOMACS) 8 (1)
(1998) 3–30.
																													*/

#define RNG32

#define MT19937_FLOAT_MULTI	  2.3283064365386962890625e-10f
#define MT19937_DOUBLE2_MULTI 2.3283064365386962890625e-10
#define MT19937_DOUBLE_MULTI  5.4210108624275221700372640e-20

#define MT19937_N		   624
#define MT19937_M		   397
#define MT19937_MATRIX_A   0x9908b0df /* constant vector a */
#define MT19937_UPPER_MASK 0x80000000 /* most significant w-r bits */
#define MT19937_LOWER_MASK 0x7fffffff /* least significant r bits */

/**
State of MT19937 RNG.
*/
typedef struct {
	uint32_t mt[MT19937_N]; /* the array for the state vector  */
	int mti;
} mt19937_state;

/**
Generates a random 32-bit unsigned integer using MT19937 RNG.

@param state State of the RNG to use.
*/
#define mt19937_uint(state) _mt19937_uint(&state)
uint32_t _mt19937_uint(mt19937_state *state) {
	uint32_t y;
	uint32_t mag01[2] = {0x0, MT19937_MATRIX_A};
	/* mag01[x] = x * MT19937_MATRIX_A  for x=0,1 */

	if (state->mti < MT19937_N - MT19937_M) {
		y = (state->mt[state->mti] & MT19937_UPPER_MASK) |
			(state->mt[state->mti + 1] & MT19937_LOWER_MASK);
		state->mt[state->mti] = state->mt[state->mti + MT19937_M] ^ (y >> 1) ^ mag01[y & 0x1];
	} else if (state->mti < MT19937_N - 1) {
		y = (state->mt[state->mti] & MT19937_UPPER_MASK) |
			(state->mt[state->mti + 1] & MT19937_LOWER_MASK);
		state->mt[state->mti] =
		  state->mt[state->mti + (MT19937_M - MT19937_N)] ^ (y >> 1) ^ mag01[y & 0x1];
	} else {
		y = (state->mt[MT19937_N - 1] & MT19937_UPPER_MASK) | (state->mt[0] & MT19937_LOWER_MASK);
		state->mt[MT19937_N - 1] = state->mt[MT19937_M - 1] ^ (y >> 1) ^ mag01[y & 0x1];
		state->mti				 = 0;
	}
	y = state->mt[state->mti++];

	/* Tempering */
	y ^= (y >> 11);
	y ^= (y << 7) & 0x9d2c5680;
	y ^= (y << 15) & 0xefc60000;
	y ^= (y >> 18);

	return y;
}
/**
Generates a random 32-bit unsigned integer using MT19937 RNG.

This is alternative implementation of MT19937 RNG, that generates 32 values in single call.

@param state State of the RNG to use.
*/
#define mt19937_loop_uint(state) _mt19937_loop_uint(&state)
uint32_t _mt19937_loop_uint(mt19937_state *state) {
	uint32_t y;
	uint32_t mag01[2] = {0x0, MT19937_MATRIX_A};
	/* mag01[x] = x * MT19937_MATRIX_A  for x=0,1 */

	if (state->mti >= MT19937_N) {
		int kk;

		for (kk = 0; kk < MT19937_N - MT19937_M; kk++) {
			y = (state->mt[kk] & MT19937_UPPER_MASK) | (state->mt[kk + 1] & MT19937_LOWER_MASK);
			state->mt[kk] = state->mt[kk + MT19937_M] ^ (y >> 1) ^ mag01[y & 0x1];
		}
		for (; kk < MT19937_N - 1; kk++) {
			y = (state->mt[kk] & MT19937_UPPER_MASK) | (state->mt[kk + 1] & MT19937_LOWER_MASK);
			state->mt[kk] = state->mt[kk + (MT19937_M - MT19937_N)] ^ (y >> 1) ^ mag01[y & 0x1];
		}
		y = (state->mt[MT19937_N - 1] & MT19937_UPPER_MASK) | (state->mt[0] & MT19937_LOWER_MASK);
		state->mt[MT19937_N - 1] = state->mt[MT19937_M - 1] ^ (y >> 1) ^ mag01[y & 0x1];

		state->mti = 0;
	}

	y = state->mt[state->mti++];

	/* Tempering */
	y ^= (y >> 11);
	y ^= (y << 7) & 0x9d2c5680;
	y ^= (y << 15) & 0xefc60000;
	y ^= (y >> 18);

	return y;
}

/**
Seeds MT19937 RNG.

@param state Variable, that holds state of the generator to be seeded.
@param seed Value used for seeding. Should be randomly generated for each instance of generator
(thread).
*/
void mt19937_seed(mt19937_state *state, uint32_t s) {
	state->mt[0] = s;
	uint32_t mti;
	for (mti = 1; mti < MT19937_N; mti++) {
		state->mt[mti] = 1812433253 * (state->mt[mti - 1] ^ (state->mt[mti - 1] >> 30)) + mti;

		/* See Knuth TAOCP Vol2. 3rd Ed. P.106 for multiplier. */
		/* In the previous versions, MSBs of the seed affect   */
		/* only MSBs of the array mt19937[].                        */
		/* 2002/01/09 modified by Makoto Matsumoto             */
	}
	state->mti = mti;
}

/**
Generates a random 64-bit unsigned integer using MT19937 RNG.

@param state State of the RNG to use.
*/
#define mt19937_ulong(state) ((((uint64_t)mt19937_uint(state)) << 32) | mt19937_uint(state))

/**
Generates a random float using MT19937 RNG.

@param state State of the RNG to use.
*/
#define mt19937_float(state) (mt19937_uint(state) * MT19937_FLOAT_MULTI)

/**
Generates a random double using MT19937 RNG.

@param state State of the RNG to use.
*/
#define mt19937_double(state) (mt19937_ulong(state) * MT19937_DOUBLE_MULTI)

template<typename Destination, typename Source>
__global__ void fillArray(size_t elements, Destination *dst, Source value) {
	const size_t kernelIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (kernelIndex < elements) { dst[kernelIndex] = value; }
}

void print_binary_16bit(int number) {
    int i;
    for (i = 15; i >= 0; i--) {
        printf((number & (1 << i)) ? "1" : "0");
    }
	printf("\n");
}

template<typename T, typename Lower, typename Upper>
__global__ void fillRandom(T *data, int64_t elements, Lower lower, Upper upper, int64_t *seeds,
						   int64_t numSeeds) {
	int64_t gid		  = blockDim.x * blockIdx.x + threadIdx.x;
	int64_t seedIndex = gid % numSeeds;
	mt19937_state state;
	mt19937_seed(&state, seeds[seedIndex]);

	for (int64_t i = gid; i < elements; i += blockDim.x * gridDim.x) {
		data[i] = (T)(mt19937_double(state) * (upper - lower) + lower);
	}

	// Change the seed for the next thread
	seeds[seedIndex] = mt19937_ulong(state);
}

template<typename T, typename Lower, typename Upper>
__global__ void fillRandomHalf(T *data, int64_t elements, Lower lower, Upper upper, int64_t *seeds,
						   int64_t numSeeds) {
	int64_t gid		  = blockDim.x * blockIdx.x + threadIdx.x;
	int64_t seedIndex = gid % numSeeds;
	mt19937_state state;
	mt19937_seed(&state, seeds[seedIndex]);

	for (int64_t i = gid; i < elements; i += blockDim.x * gridDim.x) {
		float lowerF = (float)lower;
		float upperF = (float)upper;
		data[i] = (T)(mt19937_float(state) * (upperF - lowerF) + lowerF);
	}

	// Change the seed for the next thread
	seeds[seedIndex] = mt19937_ulong(state);
}

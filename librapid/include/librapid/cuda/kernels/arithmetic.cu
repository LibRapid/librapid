#include "hip/hip_runtime.h"
#include "kernelHelper.hpp"
#include <stdint.h>
#include <stdio.h>

template<typename Destination, typename LHS, typename RHS>
__global__ void addArrays(size_t elements, Destination *dst, LHS *lhs, RHS *rhs) {
	const size_t kernelIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (kernelIndex < elements) { dst[kernelIndex] = lhs[kernelIndex] + rhs[kernelIndex]; }
}

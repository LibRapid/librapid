#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <stdexcept>
#include <chrono>

#include <librapid/config.hpp>
#include <librapid/cuda/backend.cuh>

static const char *getCublasErrorEnum_(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:return "HIPBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:return "HIPBLAS_STATUS_UNKNOWN";
    }

    return "UNKNOWN ERROR";
}

inline void cublasSafeCall_(hipblasStatus_t err, const char *file, const int line)
{
    if (HIPBLAS_STATUS_SUCCESS != err)
		throw std::runtime_error("cuBLAS function failed at line "
		+ std::to_string(line) + ", file " + std::string(file) + ": " +
		getCublasErrorEnum_(err));
}

inline void cudaSafeCall_(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
	throw std::runtime_error("CUDA function failed at line "
	+ std::to_string(line) + ", file " + std::string(file) + ": " + 
	hipGetErrorString(err));
}

__global__ void print_something_kernel(int x)
{
	printf("Here is something: %i\n", x);
}

template<typename A, typename B, typename C>
__global__ void librapid_cuda_trivial_binary(const A *a, const B *b, C *c,
											 const lr_int stride_a,
											 const lr_int stride_b,
											 const lr_int stride_c,
											 LAMBDA op)
{

}

void print_something(int x)
{
	print_something_kernel<<<1, 1>>>(x);
	hipDeviceSynchronize();
}

void *librapid_cuda_malloc(size_t bytes)
{
	void *res;
	cudaSafeCall(hipMalloc(&res, bytes));
	return res;
}

void librapid_cuda_free(void *data)
{
	cudaSafeCall(hipFree(data));
}

void librapid_cuda_memcpy(void *dst, void *src, size_t bytes, librapid_cuda_memcpykind kind)
{
	cudaSafeCall(hipMemcpy(dst, src, bytes, (hipMemcpyKind) kind));
}

template<typename A, typename B, typenaem C>
void librapid_cuda_binary_op(const A *a, const B *b, C *c,
							 bool trivial_a, bool trivial_b, bool trivial_c,
							 const lr_int[LIBRAPID_MAX_DIMS] stride_a,
							 const lr_int[LIBRAPID_MAX_DIMS] stride_b,
							 const lr_int[LIBRAPID_MAX_DIMS] stride_c)
{
	cudaSafeCall(hipDeviceSynchronize());

	// Check for simple strides
	if (trivial_a && trivial_b && trivial_c)
	{
		// Use vector add

	}
}
